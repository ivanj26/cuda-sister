#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

__global__ void vector_add(int *out, int *a, int *b, int n){
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < n; i+=stride) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    int n = 10;
    int *a, *b, *out, *dev_a, *dev_b, *dev_out;

    a = (int*) malloc(n * sizeof(int));
    b = (int*) malloc(n * sizeof(int));
    out = (int*) malloc(n * sizeof(int));

    for (int i = 0; i < n; i++) {
        a[i] = 1;
        b[i] = 2;
    }

    // Allocate device memory
    hipMalloc((void**)&dev_a, sizeof(float) * n);
    hipMalloc((void**)&dev_b, sizeof(float) * n);
    hipMalloc((void**)&dev_out, sizeof(float) * n);

    // Transfer data from host to device memory
    hipMemcpy(dev_a, a, sizeof(float) * n, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(float) * n, hipMemcpyHostToDevice);

    // Executing kernel
    vector_add<<<1,256>>>(dev_out, dev_a, dev_b, n);

    // Transfer data back to host memory
    hipMemcpy(out, dev_out, sizeof(float) * n, hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++) {
        printf("%d\n", out[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_out);

    delete[] a;
    delete[] out;
    delete[] b;

    return EXIT_SUCCESS;
}
